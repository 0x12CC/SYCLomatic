// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test6_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test6_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test6_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test6_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test6_out

// CHECK: 32
// TEST_FEATURE: LapackUtils_getrf


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  double2* a_z;
  int64_t* ipiv_s;
  hipsolverHandle_t handle;
  size_t device_ws_size_s;
  size_t host_ws_size_s;
  hipsolverDnParams_t params;
  void* device_ws_s;
  void* host_ws_s;
  int *info;

  hipsolverDnXgetrf(handle, params, 2, 2, HIP_R_32F, a_s, 2, ipiv_s, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  return 0;
}
